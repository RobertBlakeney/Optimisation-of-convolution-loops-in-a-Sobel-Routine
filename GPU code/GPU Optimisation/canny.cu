#include "hip/hip_runtime.h"
#include "canny.cuh"
//19 refs to N and M
//unsigned char filt[N][M], gradient[N][M], grad2[N][M], edgeDir[N][M]; //5 3 0 2
unsigned char gaussianMask[5][5];
signed char GxMask[3][3], GyMask[3][3];
__constant__ int d_GxMask[3][3], d_GyMask[3][3];
int width, height = 0;
float fulltimeG = 0;
float fulltimeS = 0;

// use of high res clock for accurate timing
typedef std::chrono::high_resolution_clock timer;

int main() {

	int i, j;
	const char* inList[] = { "input/100x100.pgm", "input/256x256.pgm", "input/512x512.pgm", "input/1024x1024.pgm", "input/2048x2048.pgm", "input/4096x4096.pgm" };
	const char* outListG[] = { "output/outG.pgm", "output/out2G.pgm", "output/out3G.pgm", "output/out4G.pgm", "output/out5G.pgm", "output/out6G.pgm" };
	const char* outListS[] = { "output/outS.pgm", "output/out2S.pgm", "output/out3S.pgm", "output/out4S.pgm", "output/out5S.pgm", "output/out6S.pgm" };

	image_detection(inList, outListG, outListS);

	long long int pixelS = imgNo * ((width - 2) * (height - 2));

	double ppsS = pixelS / fulltimeS;
	float GppsS = ppsS / 1000000000;

	cout << "\nTotal time to process sobel mask: " << fulltimeS << endl;

	cout << "\nGpps of sobel mask: " << GppsS << endl;

	system("pause");
	return 0;
}

__global__ void SobelGPU(unsigned char* f, unsigned char* g, int width, int height) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row >= 1 && row < height - 1 && col >= 1 && col < width - 1) {
		int Gx = 0;
		int Gy = 0;

		for (int rowOffset = -1; rowOffset <= 1; rowOffset++) {
			for (int colOffset = -1; colOffset <= 1; colOffset++) {

				Gx += f[(row + rowOffset) * width + (col + colOffset)] * d_GxMask[rowOffset + 1][colOffset + 1];
				Gy += f[(row + rowOffset) * width + (col + colOffset)] * d_GyMask[rowOffset + 1][colOffset + 1];
			}
		}

		int grad = (int)sqrtf(Gx * Gx + Gy * Gy);
		g[row * width + col] = (unsigned char)grad;
	}


}

void GaussianBlur() {

	int i, j;
	unsigned int    row, col;
	int rowOffset;
	int colOffset;
	int newPixel;

	unsigned char temp;


	/* Declare Gaussian mask */
	gaussianMask[0][0] = 2;

	gaussianMask[0][1] = 4;
	gaussianMask[0][2] = 5;
	gaussianMask[0][3] = 4;
	gaussianMask[0][4] = 2;

	gaussianMask[1][0] = 4;
	gaussianMask[1][1] = 9;
	gaussianMask[1][2] = 12;
	gaussianMask[1][3] = 9;
	gaussianMask[1][4] = 4;

	gaussianMask[2][0] = 5;
	gaussianMask[2][1] = 12;
	gaussianMask[2][2] = 15;
	gaussianMask[2][3] = 12;
	gaussianMask[2][4] = 5;

	gaussianMask[3][0] = 4;
	gaussianMask[3][1] = 9;
	gaussianMask[3][2] = 12;
	gaussianMask[3][3] = 9;
	gaussianMask[3][4] = 4;

	gaussianMask[4][0] = 2;
	gaussianMask[4][1] = 4;
	gaussianMask[4][2] = 5;
	gaussianMask[4][3] = 4;
	gaussianMask[4][4] = 2;

	/*---------------------- Gaussian Blur ---------------------------------*/
	for (row = 2; row < width - 2; row++) {
		for (col = 2; col < height - 2; col++) {
			newPixel = 0;
			for (rowOffset = -2; rowOffset <= 2; rowOffset++) {
				for (colOffset = -2; colOffset <= 2; colOffset++) {
					newPixel += frame1[row + rowOffset][col + colOffset] * gaussianMask[2 + rowOffset][2 + colOffset];  // 2 ops
				}
			}
			f[(row * width) + col] = (unsigned char*)(newPixel / 159);  // 1 ops
		}
	}


}


void Sobel() {


	int i, j;
	unsigned int    row, col;
	int rowOffset;
	int colOffset;
	int Gx;
	int Gy;
	float thisAngle;
	int newAngle = 0;
	int newPixel;

	unsigned char temp;




	/* Declare Sobel masks */
	GxMask[0][0] = -1; GxMask[0][1] = 0; GxMask[0][2] = 1;
	GxMask[1][0] = -2; GxMask[1][1] = 0; GxMask[1][2] = 2;
	GxMask[2][0] = -1; GxMask[2][1] = 0; GxMask[2][2] = 1;

	GyMask[0][0] = -1; GyMask[0][1] = -2; GyMask[0][2] = -1;
	GyMask[1][0] = 0; GyMask[1][1] = 0; GyMask[1][2] = 0;
	GyMask[2][0] = 1; GyMask[2][1] = 2; GyMask[2][2] = 1;

	/*---------------------------- Determine edge directions and gradient strengths -------------------------------------------*/
	for (row = 1; row < width - 1; row++) {
		for (col = 1; col < height - 1; col++) {

			Gx = 0;
			Gy = 0;

			/* Calculate the sum of the Sobel mask times the nine surrounding pixels in the x and y direction */
			for (rowOffset = -1; rowOffset <= 1; rowOffset++) {
				for (colOffset = -1; colOffset <= 1; colOffset++) {

					Gx += (int)f[(row + rowOffset) * width +(col + colOffset)] * GxMask[rowOffset + 1][colOffset + 1]; // 2 ops
					Gy += (int)f[(row + rowOffset) * width + (col + colOffset)] * GyMask[rowOffset + 1][colOffset + 1]; // 2 ops
				}
			}
			int val = sqrt(Gx * Gx + Gy * Gy);
			g[(row * width) + col] = (unsigned char*)val;

			thisAngle = (((atan2(Gx, Gy)) / 3.14159) * 180.0);

			/* Convert actual edge direction to approximate value */
			if (((thisAngle >= -22.5) && (thisAngle <= 22.5)) || (thisAngle >= 157.5) || (thisAngle <= -157.5))
				newAngle = 0;
			else if (((thisAngle > 22.5) && (thisAngle < 67.5)) || ((thisAngle > -157.5) && (thisAngle < -112.5)))
				newAngle = 45;
			else if (((thisAngle >= 67.5) && (thisAngle <= 112.5)) || ((thisAngle >= -112.5) && (thisAngle <= -67.5)))
				newAngle = 90;
			else if (((thisAngle > 112.5) && (thisAngle < 157.5)) || ((thisAngle > -67.5) && (thisAngle < -22.5)))
				newAngle = 135;


			//eD[row][col] = newAngle;
		}
	}

}


int image_detection(const char* inn[], const char* out1[], const char* out2[]) {


	int i, j;
	unsigned int row, col;
	int rowOffset;
	int colOffset;
	int Gx;
	int Gy;
	float thisAngle;
	int newAngle;
	int newPixel;
	unsigned char* d_f, * d_g;


	unsigned char temp;

	//We run this part of the code to get the dimensions of the image to resize arrays for the program to work
	FILE* finput;
	auto data = openfile(inn[selImg], &finput);
	height = data.first;
	width = data.second;

	f = (unsigned char**)malloc(width * height * sizeof(unsigned char*));

	g = (unsigned char**)malloc(width * height * sizeof(unsigned char*));

	//creating device arrays
	hipMalloc((void**)&d_f, width * height * sizeof(unsigned char));
	hipMalloc((void**)&d_g, width * height * sizeof(unsigned char));

	int GxMask[3][3] = {{ -1, 0, 1 }, { -2, 0, 2 }, { -1, 0, 1 }};
	int GyMask[3][3] = {{ -1, -2, -1 }, { 0, 0, 0 }, { 1, 2, 1 }};

	hipMemcpyToSymbol(HIP_SYMBOL(d_GxMask), GxMask, 3 * 3 * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_GyMask), GyMask, 3 * 3 * sizeof(int));

	//consider using realloc for better dynamic us of memory
	frame1 = (unsigned char**)malloc(width * sizeof(unsigned char*));
	if (frame1 == NULL) { printf("\nerror with malloc fr"); return -1; }
	for (i = 0; i < width; i++) {
		frame1[i] = (unsigned char*)malloc(height * sizeof(unsigned char));
		if (frame1[i] == NULL) { printf("\nerror with malloc fr"); return -1; }
	}


	//create the image
	print = (unsigned char**)malloc(width * sizeof(unsigned char*));
	if (print == NULL) { printf("\nerror with malloc fr"); return -1; }
	for (i = 0; i < width; i++) {
		print[i] = (unsigned char*)malloc(height * sizeof(unsigned char));
		if (print[i] == NULL) { printf("\nerror with malloc fr"); return -1; }
	}

	//initialize the image
	for (i = 0; i < width; i++)
		for (j = 0; j < height; j++)
			print[i][j] = 0;

	read_image(inn[selImg], frame1);

	GaussianBlur();


	for (i = 0; i < width; i++)
		for (j = 0; j < height; j++)
			print[i][j] = (unsigned char)f[(i * width) + j];

	write_image(out1[selImg], print);



	hipMemcpy(d_f, f, width * height * sizeof(unsigned char), hipMemcpyHostToDevice);

	dim3 dimBlock(16, 16);
	dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (height + dimBlock.y - 1) / dimBlock.y);

	auto ss = timer::now();

	for (int i = 0; i < imgNo; ++i) {
		Sobel();
		//SobelGPU << <dimGrid, dimBlock >> > (d_f, d_g, width, height);
	}
	
	auto es = timer::now();

	hipMemcpy(g, d_g, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost);

	hipFree(d_f);
	hipFree(d_g);



	/* write gradient to image*/

	for (i = 0; i < width; i++)
		for (j = 0; j < height; j++)
			print[i][j] = (unsigned char)g[(i * width) + j];

	write_image(out2[selImg], print);

	auto timeS = duration_cast<microseconds>(es - ss);
	float fTimeS = (float)timeS.count() / 1000000;
	fulltimeS += fTimeS;
	//cout << endl << "Time: " << fTimeG or fTimeS << "\n" << endl; //To check time of individual image

	for (i = 0; i < width; i++)
		free(frame1[i]);
	free(frame1);



	for (i = 0; i < width; i++)
		free(print[i]);
	free(print);

	free(f);
	free(g);


	return 0;

}


#pragma region fileMan




void read_image(const char filename[], unsigned char** image) // 0 ops
{
	int inint = -1;
	int c;
	FILE* finput;
	int i, j;

	printf("  Reading image from disk (%s)...\n", filename);
	//finput = NULL;
	openfile(filename, &finput);


	for (j = 0; j < width; j++)
		for (i = 0; i < height; i++) {
			c = getc(finput);


			image[j][i] = (unsigned char)c;
		}



	/* for (j=0; j<N; ++j)
	   for (i=0; i<M; ++i) {
		 if (fscanf(finput, "%i", &inint)==EOF) {
		   fprintf(stderr,"Premature EOF\n");
		   exit(-1);
		 } else {
		   image[j][i]= (unsigned char) inint; //printf("\n%d",inint);
		 }
	   }*/



	fclose(finput);
}





void write_image(const char* filename, unsigned char** image) //0 ops
{
	FILE* foutput;
	errno_t err;
	int i, j;


	printf("  Writing result to disk (%s)...\n", filename);
	if ((err = fopen_s(&foutput, filename, "wb")) != NULL) {
		printf("Unable to open file %s for writing\n", filename);
		exit(-1);
	}

	fprintf(foutput, "P2\n");
	fprintf(foutput, "%d %d\n", height, width);
	fprintf(foutput, "%d\n", 255);

	for (j = 0; j < width; ++j) {
		for (i = 0; i < height; ++i) {
			fprintf(foutput, "%3d ", image[j][i]);
			if (i % 32 == 31) fprintf(foutput, "\n");
		}
		if (height % 32 != 0) fprintf(foutput, "\n");
	}
	fclose(foutput);


}


std::pair<int, int> openfile(const char* filename, FILE** finput) //3 ops
{
	int x0, y0;
	errno_t err;
	char header[255];
	int aa;

	if ((err = fopen_s(finput, filename, "rb")) != NULL) {
		printf("Unable to open file %s for reading\n", filename);
		exit(-1);
	}

	aa = fscanf_s(*finput, "%s", header, 20);

	/*if (strcmp(header,"P2")!=0) {
	   fprintf(stderr,"\nFile %s is not a valid ascii .pgm file (type P2)\n",
			   filename);
	   exit(-1);
	 }*/

	x0 = getint(*finput);
	y0 = getint(*finput);

	//if ((x0 != M) || (y0 != N)) {
	//	printf("Image dimensions do not match: %ix%i expected\n", N, M);
	//	exit(-1);
	//}

	//x0 = getint(*finput); /* read and throw away the range info */
	return { x0, y0 };
}


int getint(FILE* fp) /* adapted from "xv" source code */
{
	int c, i, firstchar, garbage;

	/* note:  if it sees a '#' character, all characters from there to end of
	   line are appended to the comment string */

	   /* skip forward to start of next number */
	c = getc(fp);
	while (1) {
		/* eat comments */
		if (c == '#') {
			/* if we're at a comment, read to end of line */
			char cmt[256], * sp;

			sp = cmt;  firstchar = 1;
			while (1) {
				c = getc(fp);
				if (firstchar && c == ' ') firstchar = 0;  /* lop off 1 sp after # */
				else {
					if (c == '\n' || c == EOF) break;
					if ((sp - cmt) < 250) *sp++ = c;
				}
			}
			*sp++ = '\n';
			*sp = '\0';
		}

		if (c == EOF) return 0;
		if (c >= '0' && c <= '9') break;   /* we've found what we were looking for */

		/* see if we are getting garbage (non-whitespace) */
		if (c != ' ' && c != '\t' && c != '\r' && c != '\n' && c != ',') garbage = 1;

		c = getc(fp);
	}

	/* we're at the start of a number, continue until we hit a non-number */
	i = 0;
	while (1) {
		i = (i * 10) + (c - '0');
		c = getc(fp);
		if (c == EOF) return i;
		if (c < '0' || c>'9') break;
	}
	return i;
}
#pragma endregion
